#include "hip/hip_runtime.h"
#include "yolov7face_postprocess.h"
#define MAX_LANDMARK 20
static __device__ void yolov7face_affine_project(float* matrix, float x, float y, float* ox, float* oy)
{
    *ox = matrix[0] * x + matrix[1] * y + matrix[2];
    *oy = matrix[3] * x + matrix[4] * y + matrix[5];
}

static __global__ void yolov7face_decode_kernel(float* predict,int NUM_BOX_ELEMENT, int num_bboxes, int num_classes,int ckpt, float confidence_threshold, float* invert_affine_matrix, float* parray, int max_objects)
{  

    //printf("NUM_BOX_ELEMENT:%d,num_bboxes:%d,num_classes:%d,ckpt%d,confidence_threshold%f,max_objects%d",NUM_BOX_ELEMENT,num_bboxes,num_classes,ckpt,confidence_threshold,max_objects);
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= num_bboxes) //25200
    {
        return;
    }
    float*pitem= predict+(5 + num_classes+ckpt*3) * position;//每个线程处理一个人脸的21个参数
    float objectness = pitem[4];                             //置信度
    if(objectness < confidence_threshold)                    //小于该置信度的舍弃
    {
        return;
    }
    
    //多类别判断置信度和类别,找到置信度最大的类别
    float*class_confidence = pitem + 5;                     //第一个类别置信度
    float confidence= *class_confidence++;                  //先++，后*，第二个类别置信度
    int label= 0;
    for(int i = 1; i < num_classes; ++i, ++class_confidence)
    {
        if(*class_confidence > confidence)
        {
            confidence = *class_confidence;
            label = i;
        }
    }
    confidence *= objectness;                               //最终的置信度是类别置信度*object的置信度                            
    if(confidence < confidence_threshold)
    {
        return;
    }
    //判断是否超过了最大人脸框的阈值,bbox个数存放在parray[0]
    int index = atomicAdd(parray, 1);
    if(index >= max_objects)
    {
        return;
    }
    //printf("index %d max_objects %d\n", index,max_objects);
    float cx         = pitem[0];
    float cy         = pitem[1];
    float width      = pitem[2];
    float height     = pitem[3];
    
    //五个关键点
    float *landmarks = pitem+5+num_classes;
    float landmark_array[MAX_LANDMARK*2];
    for (int i = 0; i<ckpt; i++)
    {
        landmark_array[2*i]=landmarks[3*i];
        landmark_array[2*i+1]=landmarks[3*i+1];
    }

    //从cx,cy,w,h计算出(x1,y1,x2,y2)
    float left   = cx - width * 0.5f;        
    float top    = cy - height * 0.5f;
    float right  = cx + width * 0.5f;
    float bottom = cy + height * 0.5f;

    //bbox还原到原图的位置
    yolov7face_affine_project(invert_affine_matrix, left,  top,    &left,  &top);
    yolov7face_affine_project(invert_affine_matrix, right, bottom, &right, &bottom);
    //landmark还原到原图的位置
    for(int i = 0; i<ckpt; i++)
    {
        yolov7face_affine_project(invert_affine_matrix, landmark_array[2*i],landmark_array[2*i+1],&landmark_array[2*i],&landmark_array[2*i+1]); 
    }
    float* pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
    *pout_item++ = left;            //1
    *pout_item++ = top;             //2
    *pout_item++ = right;           //3
    *pout_item++= bottom;          //4
    *pout_item++ = confidence;      //5
    *pout_item++ = label;           //6
    *pout_item++ = 1;               //7 1 = keep, 0 = ignore
    for(int i = 0; i<ckpt; i++)
    {
        *pout_item++=landmark_array[2*i];
        *pout_item++=landmark_array[2*i+1];
    }
}

static __device__ float yolov7face_box_iou(float aleft, float atop, float aright, float abottom, float bleft, float btop, float bright, float bbottom)
{

    float cleft 	= max(aleft, bleft);
    float ctop 		= max(atop, btop);
    float cright 	= min(aright, bright);
    float cbottom 	= min(abottom, bbottom);
    
    float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
    if(c_area == 0.0f)
    {
        return 0.0f;
    }
    float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
    float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
    return c_area / (a_area + b_area - c_area);
}

static __global__ void yolov7face_nms_kernel(float* bboxes, int max_objects, float threshold,int NUM_BOX_ELEMENT)
{

    int position = (blockDim.x * blockIdx.x + threadIdx.x);
    //得到bbox数量
    int count = min((int)*bboxes, max_objects);
    if (position >= count) 
    {
        return;
    }
    //
    // left, top, right, bottom, confidence, class, keepflag
    float* pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
    for(int i = 0; i < count; ++i)
    {
        float* pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
        //是同一个或者不是同一类，则跳过
        if(i == position || pcurrent[5] != pitem[5]) 
        {
            continue;
        }
        //置信度大于本次并且iou大于阈值，则pcurrent[6] = 0
        if(pitem[4] >= pcurrent[4])
        {
            if(pitem[4] == pcurrent[4] && i < position)
            {
                continue;
            }
            //iou
            float iou = yolov7face_box_iou(pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3],pitem[0],pitem[1],pitem[2],pitem[3]);
            if(iou > threshold)
            {
                pcurrent[6] = 0;  // 1=keep, 0=ignore
                return;
            }
        }
    }
} 

void yolov7face_decode_kernel_invoker(float* predict, int  NUM_BOX_ELEMENT,int num_bboxes,int num_classes,int ckpt, float confidence_threshold, float* invert_affine_matrix, float* parray, int max_objects, hipStream_t stream)
{
    int block = 256;
    int  grid =  ceil((num_bboxes+block-1) / (float)block);
    
    yolov7face_decode_kernel<<<grid, block, 0, stream>>>(predict,NUM_BOX_ELEMENT, num_bboxes, num_classes,ckpt, confidence_threshold, invert_affine_matrix, parray, max_objects);
}

void yolov7face_nms_kernel_invoker(float* parray, float nms_threshold, int max_objects, hipStream_t stream,int NUM_BOX_ELEMENT)
{
    int block = max_objects<256? max_objects:256;
    int grid = ceil((max_objects+block-1) / (float)block);
    yolov7face_nms_kernel<<<grid, block, 0, stream>>>(parray, max_objects, nms_threshold,NUM_BOX_ELEMENT);
}
